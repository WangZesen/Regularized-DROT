#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>

// #include "param_qr.cuh"
#include "drot_glr.hpp"

#define CEILDIV(x, y) ((x+y-1)/y)

// auxiliary function

void group_lasso_regularizer_drot_torch_float32(
        const float *c,
        const float *p,
        const float *q,
        const int n_rows,
        const int n_cols,
        const int NGROUPS,
        const float rho,
        const float r_weight,
        const int max_iters,
        const float eps,
        const int work_size_update_x,
        float *x,
        float *a,
        float *row_sum,
        float *row_sum_1,
        float *row_sum_2,
        float *b,
        float *col_sum,
        float *col_sum_1,
        float *col_sum_2,
        float *col_obj_sum,
        float *col_norm_sum,
        float *group_sum,
        float *group_obj_sum,
        float *group_norm_sum,
        float *phi1,
        float *phi2,
        float *aux) {
    
    float step_size, objective;
    int n_iters, row_size, col_size, mat_size;
    step_size = rho / (float(n_rows) + float(n_cols));

    const float lambda = r_weight * sqrt(n_rows / NGROUPS);
    row_size = n_rows * sizeof(float);
    col_size = n_cols * sizeof(float);
    mat_size = n_rows * n_cols * sizeof(float);

    // initialization
    const float _n = float(n_rows);
    const float _m = float(n_cols);
    const float _k = float(1.0) * step_size * _n * _m - float(2.);

    const float v_phi1 = (_k + float(2)) / _n / (_m + _n);
    const float v_phi2 = (_k + float(2)) / _m / (_m + _n);
    const float v_a = (_k + float(1)) / _n;
    const float v_b = (_k + float(1)) / _m;
    const float v_alpha = _k + float(1);
    const float v_beta = 0;

    std::vector<float> c_phi1(n_rows, v_phi1);
    std::vector<float> c_phi2(n_cols, v_phi2);
    std::vector<float> c_a1(n_rows, v_a);
    std::vector<float> c_b1(n_cols, v_b);
    std::vector<float> c_alpha_gamma(2);
    c_alpha_gamma[0] = v_alpha;
    c_alpha_gamma[1] = v_beta;

    hipMemcpy(phi1, &c_phi1[0], row_size, hipMemcpyHostToDevice);
    hipMemcpy(phi2, &c_phi2[0], col_size, hipMemcpyHostToDevice);
    hipMemcpy(a, &c_a1[0], row_size, hipMemcpyHostToDevice);
    hipMemcpy(b, &c_b1[0], col_size, hipMemcpyHostToDevice);
    hipMemset(aux, 0, 5*sizeof(float));
    hipMemcpy(&aux[4], &c_alpha_gamma[0], sizeof(float), hipMemcpyHostToDevice);
    hipMemset(x, 0, mat_size);

    group_lasso_regularizer_drot<float>(
            c, p, q, n_rows, n_cols, NGROUPS, step_size, lambda,
            max_iters, eps, work_size_update_x, x, a, row_sum,
            row_sum_1, row_sum_2, b, col_sum, col_sum_1, col_sum_2,
            col_obj_sum, col_norm_sum, group_sum, group_obj_sum,
            group_norm_sum, phi1, phi2, aux, &n_iters, &objective);
}


torch::Tensor group_lasso_drot_cuda(
    torch::Tensor c,
    torch::Tensor p,
    torch::Tensor q,
    int n_class,
    float rho,
    float r_weight,
    int max_iters,
    float eps) {
    
    const int NGROUPS = n_class;
    const int n_rows = (int) p.numel();
    const int n_cols = (int) q.numel();

    TORCH_CHECK((int) c.size(0) == n_cols, "C.size(0) does not match with q.numel()");
    TORCH_CHECK((int) c.size(1) == n_rows, "C.size(1) does not match with p.numel()");

    // tensor options
    auto options = torch::TensorOptions()
        .dtype(torch::kFloat32)
        .device(torch::kCUDA)
        .requires_grad(false);

    // allocate output: transportation plan
    auto x = torch::zeros_like(c, options);

    // allocate temporary variables
    const int work_size_update_x = _gl_get_work_size_update_x(n_rows, n_cols);

    auto a = torch::empty({n_rows}, options);
    auto row_sum = torch::empty({n_rows}, options);
    auto row_sum_1 = torch::empty({n_rows * CEILDIV(n_cols, work_size_update_x)}, options);
    auto row_sum_2 = torch::empty({n_rows * CEILDIV(n_cols, work_size_update_x)}, options);
    auto b = torch::empty({n_cols}, options);
    auto col_sum = torch::empty({n_cols}, options);
    auto col_sum_1 = torch::empty({n_cols * CEILDIV(CEILDIV(n_rows, NGROUPS), UPDATE_X_BLOCK_SIZE_X) * NGROUPS}, options);
    auto col_sum_2 = torch::empty({n_cols * CEILDIV(CEILDIV(n_rows, NGROUPS), UPDATE_X_BLOCK_SIZE_X) * NGROUPS}, options);
    auto col_obj_sum = torch::empty({n_cols * CEILDIV(CEILDIV(n_rows, NGROUPS), UPDATE_X_BLOCK_SIZE_X) * NGROUPS}, options);
    auto col_norm_sum = torch::empty({n_cols * CEILDIV(CEILDIV(n_rows, NGROUPS), UPDATE_X_BLOCK_SIZE_X) * NGROUPS}, options);
    auto group_sum = torch::empty({n_cols * NGROUPS}, options);
    auto group_obj_sum = torch::empty({n_cols * NGROUPS}, options);
    auto group_norm_sum = torch::empty({n_cols * NGROUPS}, options);

    auto phi1 = torch::empty({n_rows}, options);
    auto phi2 = torch::empty({n_cols}, options);

    auto aux = torch::empty({5}, options);

    // run

    group_lasso_regularizer_drot_torch_float32(
        c.data_ptr<float>(),
        p.data_ptr<float>(),
        q.data_ptr<float>(),
        n_rows,
        n_cols,
        NGROUPS,
        rho,
        r_weight,
        max_iters,
        eps,
        work_size_update_x,
        x.data_ptr<float>(),
        a.data_ptr<float>(),
        row_sum.data_ptr<float>(),
        row_sum_1.data_ptr<float>(),
        row_sum_2.data_ptr<float>(),
        b.data_ptr<float>(),
        col_sum.data_ptr<float>(),
        col_sum_1.data_ptr<float>(),
        col_sum_2.data_ptr<float>(),
        col_obj_sum.data_ptr<float>(),
        col_norm_sum.data_ptr<float>(),
        group_sum.data_ptr<float>(),
        group_obj_sum.data_ptr<float>(),
        group_norm_sum.data_ptr<float>(),
        phi1.data_ptr<float>(),
        phi2.data_ptr<float>(),
        aux.data_ptr<float>());

    // return transportation plan
    return x;
}
